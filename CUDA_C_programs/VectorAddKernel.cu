﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define ERR_CHK(call) { gpuAssert((call), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t err, const char* file, int line, bool abort = true)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(err), file, line);
        if (abort) exit(err);
    }
}

__global__ void vecAddKernel(int* A, int* B, int* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}


int main() {

    int* d_A, * d_B, * d_C;
    int* h_A, * h_B, * h_C;
    const int n = 1024;

    h_A = (int*)malloc(n * sizeof(int));
    h_B = (int*)malloc(n * sizeof(int));
    h_C = (int*)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        h_A[i] = rand();
        h_B[i] = rand();
        h_C[i] = 0;
    }

    ERR_CHK(hipMalloc((void**)&d_A, n * sizeof(int)));
    ERR_CHK(hipMalloc((void**)&d_B, n * sizeof(int)));
    ERR_CHK(hipMalloc((void**)&d_C, n * sizeof(int)));

    ERR_CHK(hipMemcpy(d_A, h_A, n * sizeof(int), hipMemcpyHostToDevice));
    ERR_CHK(hipMemcpy(d_B, h_B, n * sizeof(int), hipMemcpyHostToDevice));


    dim3 gridSize(ceil(n / 256), 1, 1);
    dim3 blockSize(256, 1, 1);
    vecAddKernel <<< gridSize, blockSize >>> (d_A, d_B, d_C, n);
    hipError_t err = hipGetLastError();
    ERR_CHK(err);

    ERR_CHK(hipMemcpy(h_C, d_C, n * sizeof(int), hipMemcpyDeviceToHost));



    //verifying our solution
    for (int i = 0; i < n; i++) {
        if (h_A[i] + h_B[i] != h_C[i]) {
            printf("Incorrect addition");
            printf("%d + %d = %d for i = %d\n", h_A[i], h_B[i], h_C[i], i);
        }

    }
    printf("SUCCESS!!!!!!!!!!!");
    return 0;
}
